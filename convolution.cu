#include "hip/hip_runtime.h"
#include "convolution.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <thrust/device_vector.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define DIM 256
#define MASK_WIDTH 3
#define TILE_WIDTH 4
#define RADIUS 1

__constant__ float mask[MASK_WIDTH][MASK_WIDTH];

__global__ void conv2d(float* input, float* output) {
  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  
  // index of the element in the output array
  int x_o = bx * TILE_WIDTH + tx;
  int y_o = by * TILE_WIDTH + ty;

  __shared__ float N_ds[TILE_WIDTH][TILE_WIDTH];
  
  // load the tile to shared memory.
  if (0 <= x_o && x_o < DIM && 0 <= y_o && y_o < DIM)
    N_ds[ty][tx] = input[DIM * y_o + x_o];
  else
    N_ds[ty][tx] = 0;
  __syncthreads();
  
  // offset (0, 0, 0) by (-Radius, -Radius, -Radius) so that we start computing
  // at the top left corner of the mask
  int x_start = tx - RADIUS;
  int y_start = ty - RADIUS;
  float result = 0;
  
  for (int i = 0; i < MASK_WIDTH; i++)
    for (int j = 0; j < MASK_WIDTH; j++) {      
      int x_index = x_start + i;
      int y_index = y_start + j;
      
      // if the indices are within range of what we have in the tile
      // use the shared memory data
      if (   0 <= x_index && x_index < TILE_WIDTH 
          && 0 <= y_index && y_index < TILE_WIDTH)
        result += N_ds[y_index][x_index] * mask[j][i];
      // otherwise just go to global memory
      else {
        int x_global = bx * TILE_WIDTH + x_index;
        int y_global = by * TILE_WIDTH + y_index;
        
        // if we're trying to access something outside the actual matrix itself, take the value to be 0
        // (aka do nothing)
        if (   0 <= x_global && x_global < DIM 
            && 0 <= y_global && y_global < DIM)
          result += input[DIM * y_global + x_global] * mask[j][i];
      }
    } 
  
  if (x_o < DIM && y_o < DIM)
    output[DIM * y_o + x_o] = result;
  
  __syncthreads();
}


void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A)
{
  // Create a pseudo-random number generator
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);

  // Set the seed for the random number generator using the system clock
  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

  // Fill the array with random numbers on the device
  hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}


HeightmapGenerator::HeightmapGenerator(float* noiseValues) {
  output_h = (float*) malloc(DIM * DIM * sizeof(float));
  mask_h = (float*) malloc(MASK_WIDTH * MASK_WIDTH * sizeof(float));

  mask_h[0] = 1.0; mask_h[1] = 2.0; mask_h[2] = 1.0; 
  mask_h[3] = 2.0; mask_h[4] = 4.0; mask_h[5] = 2.0; 
  mask_h[6] = 1.0; mask_h[7] = 2.0; mask_h[8] = 1.0; 

  gpuErrchk(hipMalloc((void**) &input_d, DIM * DIM * sizeof(float)));
  gpuErrchk(hipMalloc((void**) &output_d, DIM * DIM * sizeof(float)));

  gpuErrchk(hipMemset(input_d, 0, DIM * DIM * sizeof(float)));

  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(mask), mask_h, MASK_WIDTH * MASK_WIDTH * sizeof(float)));
};

void HeightmapGenerator::run() {
  GPU_fill_rand(input_d, DIM, DIM);
  gpuErrchk(hipDeviceSynchronize());

  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
  dim3 dimGrid(ceil(1.0f * DIM / TILE_WIDTH), ceil(1.0f * DIM / TILE_WIDTH), 1.0f);

  conv2d<<<dimGrid, dimBlock>>>(input_d, output_d);
  gpuErrchk(hipDeviceSynchronize());

  thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(output_d);
  maxHeight = *(thrust::max_element(dev_ptr, dev_ptr + DIM * DIM));
  maxHeight = *(thrust::min_element(dev_ptr, dev_ptr + DIM * DIM));
  // thrust::device_vector<float> output_thrust;
  // output_thrust.data() = thrust::device_pointer_cast(output_d);
  // maxHeight = *(thrust::max_element(output_thrust.begin(), output_thrust.end()));
  // minHeight = *(thrust::min_element(output_thrust.begin(), output_thrust.end()));

  std::cerr << maxHeight << ", " << minHeight << std::endl;

  gpuErrchk(hipMemcpy(output_h, output_d, DIM * DIM * sizeof(float), hipMemcpyDeviceToHost));
};

HeightmapGenerator::~HeightmapGenerator() {
  free(mask_h);
  free(output_h);
  hipFree(input_d);
  hipFree(output_d);
}

float HeightmapGenerator::getMaxHeight() {
  return maxHeight;
}

float HeightmapGenerator::getMinHeight() {
  return minHeight;
}